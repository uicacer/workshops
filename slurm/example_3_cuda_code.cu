/*
 * CUDA Math Hello World - Beginner Version with Extensive Comments
 * 
 * This program demonstrates the basics of GPU programming with CUDA.
 * It launches 8 threads on the GPU, each computing the square of its thread ID.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>  // For printf() function

/*
 * GPU FUNCTION (called a "kernel")
 * The __global__ keyword tells CUDA this function runs ON THE GPU, not the CPU.
 * This function will be executed by multiple GPU threads simultaneously.
 */
__global__ void compute(int *data) {
    // Get this thread's unique ID number (0, 1, 2, 3, 4, 5, 6, or 7)
    // threadIdx.x is a built-in CUDA variable that tells each thread its ID
    int idx = threadIdx.x;
    
    // Each thread computes the square of its ID and stores it in the array
    // Thread 0 computes 0^2 = 0, thread 1 computes 1^2 = 1, etc.
    data[idx] = idx * idx;  
    
    // Print from the GPU! This shows which thread did what computation
    // Note: GPU printf might appear out of order since threads run in parallel
    printf("GPU thread %d computed %d^2 = %d\n", idx, idx, data[idx]);
}

/*
 * MAIN FUNCTION - runs on the CPU
 * This coordinates the GPU work and manages memory
 */
int main() {
    printf("Starting GPU math computation...\n");
    
    // STEP 1: Allocate memory ON THE GPU
    // We need space for 8 integers (one per GPU thread)
    int *d_data;  // 'd_' prefix means "device" (GPU) memory
    
    // cudaMalloc is like malloc(), but it allocates memory on the GPU
    // 8 * sizeof(int) = space for 8 integers
    hipMalloc(&d_data, 8 * sizeof(int));
    
    // STEP 2: Launch the GPU kernel
    // The <<<1, 8>>> syntax means:
    //   - 1 block of threads
    //   - 8 threads per block  
    //   - So total: 1 × 8 = 8 threads running in parallel
    // Each of these 8 threads will execute the compute() function simultaneously
    compute<<<1, 8>>>(d_data);
    
    // STEP 3: Wait for all GPU threads to finish
    // cudaDeviceSynchronize() makes the CPU wait until the GPU is completely done
    // Without this, the CPU might continue before the GPU finishes
    hipDeviceSynchronize();
    
    // STEP 4: Clean up GPU memory
    // cudaFree is like free(), but for GPU memory
    // Always free what you allocate!
    hipFree(d_data);
    
    printf("GPU computation completed!\n");
    return 0;
}
